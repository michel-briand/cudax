
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using std::cout;;
using std::cerr;
using std::endl;

// Shut down MPI cleanly if something goes wrong
void my_abort(int err)
{
    cout << "Test FAILED\n";
}

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
        my_abort(err); }

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1000;
  float *h_x = NULL;
  float *d_x = NULL;
  float *h_y = NULL;
  float *d_y = NULL;

  // allocate memory for arrays on the host
  h_x = (float *) malloc(N*sizeof(float));
  if (!h_x)
	  cerr << "can't allocate memory h_x" << endl;
  h_y = (float *) malloc(N*sizeof(float));
  if (!h_y)
	  cerr << "can't allocate memory h_y" << endl;

  // allocate memory for arrays on the device
  CUDA_CHECK(hipMalloc((void **)&d_x, N*sizeof(float)));
  if (!d_x)
	  cerr << "can't allocate memory d_x" << endl;

  CUDA_CHECK(hipMalloc((void **)&d_y, N*sizeof(float)));
  if (!d_y)
	  cerr << "can't allocate memory d_y" << endl;

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    h_x[i] = 1.0f;
    h_y[i] = 2.0f;
  }

  // copy arrays to device
  CUDA_CHECK(hipMemcpy(d_x, h_x, N*sizeof(float),
                             hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_y, h_y, N*sizeof(float),
                             hipMemcpyHostToDevice));

  // Run kernel on the GPU
  add<<<1, 1>>>(N, d_x, d_y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // copy array to the host
  CUDA_CHECK(hipMemcpy(h_y, d_y, N*sizeof(float),
                        hipMemcpyDeviceToHost));

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;

  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(h_y[i]-3.0f));

  cout << "Max error: " << maxError << std::endl;

  // Free memory
  free(h_x);
  free(h_y);
  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
